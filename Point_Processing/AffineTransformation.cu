#include "hip/hip_runtime.h"
//
// Created by delbek on 10/9/24.
//

#include "AffineTransformation.cuh"
#include "DeviceKernels.cuh"
#include "GPUHelpers.cuh"
#include "Vector.h"
#include <hipsolver.h>


__global__ void applyTransformation(double* matrix, unsigned* permutation, unsigned row, unsigned col)
{
    unsigned totalThreadCount = blockDim.x * gridDim.x;
    unsigned threadID = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned gridSize = row * col;
    unsigned chunkSize = (gridSize + totalThreadCount - 1) / totalThreadCount;
    unsigned start = threadID * chunkSize;
    unsigned end = min((threadID + 1) * chunkSize, gridSize);

    for (unsigned iter = start; iter < end; ++iter)
    {
        unsigned i = iter / col;
        unsigned j = iter % col;

        int new_i = matrix[0] * i + matrix[1] * j + matrix[2];
        int new_j = matrix[3] * i + matrix[4] * j + matrix[5];
        new_i = max(0, min(new_i, row - 1));
        new_j = max(0, min(new_j, col - 1));

        unsigned new_index = new_i * col + new_j;
        permutation[iter] = new_index;
    }
}

__global__ void jacobi(double* A, double* x, double* b)
{

}

void AffineTransformation::applyAffineTransformation(Image &image, const Matrix &transformationMatrix)
{
    unsigned gridSize = image.gridSize();
    unsigned* permutation = new unsigned[gridSize];
    for (int i = 0; i < gridSize; ++i) permutation[i] = i;

    unsigned* d_permutation;
    hipMalloc(&d_permutation, sizeof(unsigned) * gridSize);
    hipMemcpy(d_permutation, permutation, sizeof(unsigned) * gridSize, hipMemcpyHostToDevice);

    double* d_matrix;
    hipMalloc(&d_matrix, sizeof(double) * 3 * 3);
    hipMemcpy(d_matrix, transformationMatrix.getMatrix(), sizeof(double) * 3 * 3, hipMemcpyHostToDevice);

    int gridDimension;
    int blockDimension;
    hipOccupancyMaxPotentialBlockSizeVariableSMem(
        &gridDimension,
        &blockDimension,
        applyTransformation,
        noShared,
        0);

    applyTransformation<<<gridDimension, blockDimension>>>(d_matrix, d_permutation, image.getHeight(), image.getWidth());
    hipDeviceSynchronize();

    hipMemcpy(permutation, d_permutation, sizeof(unsigned) * gridSize, hipMemcpyDeviceToHost);
    image.applyPermutation(permutation);

    hipFree(d_permutation);
    hipFree(d_matrix);
    delete[] permutation;
}

Matrix AffineTransformation::estimateAffineTransformation(unsigned int* referencePoints, unsigned int* controlPoints, unsigned int numberOfPoints)
{
    Vector b(numberOfPoints * 2);
    for (unsigned int i = 0; i < numberOfPoints * 2; ++i)
    {
        b[i] = static_cast<double>(referencePoints[i]);
    }

    Matrix A(numberOfPoints * 2, 6);
    for (unsigned int i = 0; i < numberOfPoints; ++i)
    {
        A[i * 2 * 6 + 0] = controlPoints[i * 2];
        A[i * 2 * 6 + 1] = controlPoints[i * 2 + 1];
        A[i * 2 * 6 + 2] = 1.0;
        A[i * 2 * 6 + 3] = 0.0;
        A[i * 2 * 6 + 4] = 0.0;
        A[i * 2 * 6 + 5] = 0.0;

        A[(i * 2 + 1) * 6 + 0] = 0.0;
        A[(i * 2 + 1) * 6 + 1] = 0.0;
        A[(i * 2 + 1) * 6 + 2] = 0.0;
        A[(i * 2 + 1) * 6 + 3] = controlPoints[i * 2];
        A[(i * 2 + 1) * 6 + 4] = controlPoints[i * 2 + 1];
        A[(i * 2 + 1) * 6 + 5] = 1.0;
    }
    A = std::move(A.getColMajor());

    double* h_x = new double[6];
    double* d_A, * d_b, * d_x;

    // to be completed

    Matrix affineMatrix(3, 3);
    affineMatrix[0] = h_x[0];  // a11
    affineMatrix[1] = h_x[1];  // a12
    affineMatrix[2] = h_x[2];  // tx
    affineMatrix[3] = h_x[3];  // a21
    affineMatrix[4] = h_x[4];  // a22
    affineMatrix[5] = h_x[5];  // ty
    affineMatrix[6] = 0.0;     // last row is [0 0 1]
    affineMatrix[7] = 0.0;
    affineMatrix[8] = 1.0;

    delete[] h_x;

    return affineMatrix;
}
